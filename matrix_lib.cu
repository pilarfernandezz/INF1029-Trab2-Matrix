#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "matrix_lib.h"
#define THREADS_PER_BLOCK 256

// Kernel function to add the elements of two arrays
__global__ 
void mult_scalar(int n , float scalar, float *d_a){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x *blockDim.x;
    for(int i = index; i < n; i += stride){
       d_a[i] = d_a[i] * scalar;
    }
}

__global__
void mult_matrix(int w_a, int w_b, int h_b, int h_a, float *d_a, float *d_b, float *d_c){

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x *blockDim.x;

    int w_c = h_a, h_c = w_b;
    // Calculando a matriz resultante
    for(int i = index; i < w_c*h_c; i += stride) {
        d_c[i] = 0;
        for(int j = 0; j < h_a; j++) {
            d_c[i] += d_a[(i/h_c)*h_a + j] * d_b[j*h_b + i%h_c];
        }
    }
}

int scalar_matrix_mult(float scalar_value, struct matrix *matrix){
    long unsigned int h;
	long unsigned int w;

	h = matrix->height;
	w = matrix->width;

	if(matrix == NULL) return 0;

    int blockSize = THREADS_PER_BLOCK;
    int numBlocks = (h*w + blockSize - 1) / blockSize;
    mult_scalar<<<numBlocks, blockSize>>>(h*w,scalar_value,matrix->d_rows);

	return 1;
}

int matrix_matrix_mult(struct matrix *matrixA, struct matrix *matrixB, struct matrix *matrixC){	

	long unsigned int h_a;
	long unsigned int w_a;
	long unsigned int h_b;
	long unsigned int w_b;
	long unsigned int h_c;
	long unsigned int w_c;

    if(matrixA == NULL || matrixB == NULL|| matrixC == NULL) return 0;

    h_a = matrixA->height;
	w_a = matrixA->width;
	h_b = matrixB->height;
	w_b = matrixB->width;
	h_c = matrixC->height;
	w_c = matrixC->width;

    int blockSize = THREADS_PER_BLOCK;
    int numBlocks = (h_c*w_c + blockSize - 1) / blockSize;
    mult_matrix<<<numBlocks, blockSize>>>(w_a, w_b, h_b, h_a,matrixA->d_rows, matrixB->d_rows, matrixC->d_rows);

    return 1;
}